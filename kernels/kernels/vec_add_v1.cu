#include <stdio.h>
#include <hip/hip_runtime.h>
#include <>
#include "../include/hip/hip_runtime.h"
#include "../include/vec_add.h"

extern "C" void launch_vector_add_v1(
    const float* v1,
    const float* v2,
    float* result,
    size_t n
) {
    if (!v1 || !v2 || !result || n == 0) {
        fprintf(stderr, "Invalid parameters\n");
        return;
    }

    float *dev_v1, *dev_v2, *dev_res;

    CUDA_OK(hipMalloc((void**)&dev_v1, n * sizeof(float)));
    CUDA_OK(hipMalloc((void**)&dev_v2, n * sizeof(float)));
    CUDA_OK(hipMalloc((void**)&dev_res, n * sizeof(float)));

	CUDA_OK(hipMemcpy(dev_v1, v1, n * sizeof(float), hipMemcpyHostToDevice));
	CUDA_OK(hipMemcpy(dev_v2, v2, n * sizeof(float), hipMemcpyHostToDevice));

    size_t block_size = 256;
    size_t grid_size = (n + block_size - 1) / block_size;
    vector_add<<<grid_size, block_size>>>(dev_v1, dev_v2, dev_res, n);

    CUDA_OK(hipMemcpy(result, dev_res, n * sizeof(float), hipMemcpyDeviceToHost));

    CUDA_OK(hipFree(dev_res));
	CUDA_OK(hipFree(dev_v2));
    CUDA_OK(hipFree(dev_v1));
}
