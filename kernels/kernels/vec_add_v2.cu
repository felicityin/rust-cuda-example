#include <stdio.h>
#include <hip/hip_runtime.h>
#include <>
#include "../include/hip/hip_runtime.h"
#include "../include/vec_add.h"

extern "C" const char* launch_vector_add_v2(
    const float* v1,
    const float* v2,
    float* result,
    size_t n
) {
    if (!v1 || !v2 || !result || n == 0) {
        fprintf(stderr, "Invalid parameters\n");
    }

    float *dev_v1, *dev_v2, *dev_res;

    CUDA_OK(hipMalloc((void**)&dev_v1, n * sizeof(float)));
    CUDA_OK(hipMalloc((void**)&dev_v2, n * sizeof(float)));
    CUDA_OK(hipMalloc((void**)&dev_res, n * sizeof(float)));

	CUDA_OK(hipMemcpy(dev_v1, v1, n * sizeof(float), hipMemcpyHostToDevice));
	CUDA_OK(hipMemcpy(dev_v2, v2, n * sizeof(float), hipMemcpyHostToDevice));

    const char* res = launchKernel(vector_add, n, 0, dev_v1, dev_v2, dev_res, n);

    CUDA_OK(hipMemcpy(result, dev_res, n * sizeof(float), hipMemcpyDeviceToHost));

    CUDA_OK(hipFree(dev_res));
	CUDA_OK(hipFree(dev_v2));
    CUDA_OK(hipFree(dev_v1));

    return res;
}
